
#include <hip/hip_runtime.h>
// kernel function to add two vectors

__global__ void vecAddKernel(float* A, float* B, float* C, int n) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < n) { 
        C[i] = A[i] + B[i];
    } 
}

void vecAdd(float* A, float* B, float* C, int n) { 
    float *A_d, *B_d, *C_d; 
    int size = n * sizeof(float); 
    
    // allocate memory on the device
    hipMalloc((void **) &A_d, size); 
    hipMalloc((void **) &B_d, size); 
    hipMalloc((void **) &C_d, size); 
    
    // copy input data from host to device
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice); 
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice); 
    
    // launch the kernel
    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n); 

    // copy result from device to host
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost); 
    
    // free memory on the device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
} 